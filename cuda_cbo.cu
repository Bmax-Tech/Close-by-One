
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <hip/driver_types.h>
#include <errno.h>
#include <host_defines.h>
#include <stdbool.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_vector_types.h>

// CUDA device Max Thread Blocks
int THREADS_PER_BLOCK = 32;

clock_t start, end;
extern int errno; // globally holds the error no
int data_size; // holds the data set size read from .cxt file
int attribute_size; // holds the attribute size read from .cxt file
int *cross_table; // holds data set matrix cross table from .cxt file
int concept_count = 0; // holds generated concept count

// define concept_t for hold each concept objects and attribute sets
typedef struct {
	int *objects;
	int *attributes;
} concept_t;

concept_t *concept_latice; // holds main concept latice, generated output

// local functions
void loadData(char *file_path);
void buildInitialConcept(int obj[], int attr[]);
void computeConceptFrom(int *obj, int *attr, int attr_index);
void processConcept(int *obj, int *attr);
bool checkAttribute(int j, int *attr);
void makeExtent(int *extent, int *obj, int attr_index);
void makeIntent(int *intent, int *extent, int attr_index);
bool canonicity(int *attr, int *intent, int attr_index);
int getBlocksPerGrid(int size);

// CUDA Device variables
size_t pitch;
int *d_cross_table; // holds device cross table

// CUDA Kernels
__global__ void processConceptKernel(int *set_1, int *set_2, int size);
__global__ void makeExtentKernel(int *extent, int *obj, int *cross_objects,
		int attr_index, int rows, int cols);
__global__ void makeIntentKernelOnExtentEmpty(int *extent, int *empty_count,
		int size);
__global__ void makeIntentKernel(int *intent, int *extent, int *cross_objects,
		int empty_count, int rows, int cols);
__global__ void canonicityKernel(int *set_1, int *set_1_c, int *set_2,
		int *set_2_c, int *attr, int *intent, int cols);
__global__ void canonicityKernelSetCheck(int *set_1, int *set_2, bool *status,
		int cols);

int main(int argc, char *argv[]) {
	loadData(argv[1]); // read data from file path

	// --------------------------------------------------------------------------------------------------------------
	// allocate device cross table
	hipError_t cudaStatus;
	cudaStatus = hipMallocPitch(&d_cross_table, &pitch,
			sizeof(int) * attribute_size, data_size);
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed");
	}
	// copy data to device - cross_table
	cudaStatus = hipMemcpy(d_cross_table, cross_table,
			attribute_size * sizeof(int) * data_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize(); // force host tp pause until the copying is done
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy2D failed");
	}
	// --------------------------------------------------------------------------------------------------------------

	int ini_obj[data_size]; // initial concept object list
	int ini_attr[attribute_size]; // initial concept attribute list
	buildInitialConcept(ini_obj, ini_attr); // make object and attribute list

	concept_latice = (concept_t *) malloc(
			data_size * attribute_size * sizeof(concept_t *)); // allocate memory on concept latice

	start = clock(); // start timing
	computeConceptFrom(ini_obj, ini_attr, 0); // invoke Close-by-One
	end = clock(); // stop timing

	printf("\nTotal Concepts : %d\n\n", concept_count);
	printf("execution time : %f seconds\n\n",
			((double) (end - start) / CLOCKS_PER_SEC));

	// --------------------------------------------------------------------------------------------------------------
	hipFree(d_cross_table);
	// --------------------------------------------------------------------------------------------------------------

	free(cross_table);
	free(concept_latice);

	return 0;
}

// load data set file from given location
void loadData(char *file_path) {
	int errnum;
	FILE *file;
	if ((file = fopen(file_path, "rt")) == NULL) {
		errnum = errno;
		fprintf(stderr, "Value of errno: %d\n", errno);
		perror("Error printed by perror");
		fprintf(stderr, "Error opening file: %s\n", strerror(errnum));
	} else {
		printf("\n~~~ Dataset Cross Table ~~~\n\n");

		char buffer[256];
		int line_count = 0;
		int obj_count = 0;
		int atr_count = 0;
		while (fgets(buffer, sizeof(buffer), file)) {
			// process lines
			if (buffer[0] == '\n') {
				// new line found
			} else {
				// skip first character on the .cxt file
				if (line_count != 0) {
					if (line_count == 1) {
						// data size found
						data_size = atoi(buffer);
					} else if (line_count == 2) {
						// attribute size found
						attribute_size = atoi(buffer);
						cross_table = (int *) malloc(
								sizeof(int) * attribute_size * data_size); // allocate cross table matrix
					} else if (line_count > (2 + data_size)
							&& line_count <= (2 + data_size + attribute_size)) {
						// read attributes
						atr_count++;
						obj_count = 0; // reset obj count
					} else if (line_count > (2 + data_size + attribute_size)) {
						// read cross table
						int x;
						for (x = 0; x < attribute_size; x++) {
							// check attribute present or not
							if (buffer[x] == 'X') {
								cross_table[obj_count * attribute_size + x] = 1; // assign one when 'X'
							} else {
								cross_table[obj_count * attribute_size + x] = 0; // assign zero when '.'
							}
							printf("%d",
									cross_table[obj_count * data_size + x]);
						}
						printf("\n");
						obj_count++;
					}
				}
				line_count++;
			}
		}
		fclose(file);
		printf("\n");
	}
}

// build up initial concept
// out: objects, attributes
void buildInitialConcept(int obj[], int attr[]) {
	int i;
	int a;
	/**
	 * assign objects
	 * pass all objects into list, according to the theorem, (X)
	 */
	for (i = 0; i < data_size; i++) {
		obj[i] = i;
	}

	/**
	 * assign attributes
	 * set common attribute list for all objects on cross table (X up)
	 */
	// go through attributes
	for (a = 0; a < attribute_size; a++) {
		bool status = true;
		// go through objects
		for (i = 0; i < data_size; i++) {
			if (cross_table[i * attribute_size + a] == 0) {
				status = false;
				break;
			}
		}
		if (status) {
			// attribute available for all objects, assign 1
			attr[a] = 1;
		} else {
			attr[a] = 0;
		}
	}
}

/**
 * Close-by-One Algorithm
 *
 * input :  1. object list
 *          2. attribute list
 *          3. current attribute index
 */
void computeConceptFrom(int *obj, int *attr, int attr_index) {
	// 1. Process Concept
	processConcept(obj, attr);
	// 2. go through attribute list
	int j;
	for (j = attr_index; j < attribute_size; j++) {
		// 3. check current attribute exist or not
		if (!checkAttribute(j, attr)) {
			// 4. make extent
			int extent[data_size];
			makeExtent(extent, obj, j);
			// 5. make intent
			int intent[attribute_size];
			makeIntent(intent, extent, j);
			// 6. do canonicity test
			if (canonicity(attr, intent, j)) {
				// 7. call computeConceptFrom
				computeConceptFrom(extent, intent, (j + 1));
			}
		}
	}
}

// store concept
void processConcept(int *obj, int *attr) {
	// set objects details on concept latice
	concept_latice[concept_count].objects = (int *) malloc(
			sizeof(int) * data_size);

	// --------------------------------------------------------------------------------------------------------------
	// >>> Run CUDA Kernel <<<
	int *d_obj;
	int *d_object;
	hipMalloc((void**) &d_obj, sizeof(int) * data_size);
	hipMalloc((void**) &d_object, sizeof(int) * data_size);
	hipMemcpy(d_obj, obj, sizeof(int) * data_size, hipMemcpyHostToDevice);
	int blocksPerGrid = getBlocksPerGrid(data_size);
	processConceptKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_obj, d_object,
			data_size);
	hipMemcpy(concept_latice[concept_count].objects, d_object,
			sizeof(int) * data_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_obj);
	hipFree(d_object);
	// --------------------------------------------------------------------------------------------------------------

	// set attribute details on concept latice
	concept_latice[concept_count].attributes = (int *) malloc(
			sizeof(int) * attribute_size);

	// --------------------------------------------------------------------------------------------------------------
	// >>> Run CUDA Kernel <<<
	int *d_attr;
	int *d_attributes;
	hipMalloc((void**) &d_attr, sizeof(int) * attribute_size);
	hipMalloc((void**) &d_attributes, sizeof(int) * attribute_size);
	hipMemcpy(d_attr, attr, sizeof(int) * attribute_size,
			hipMemcpyHostToDevice);
	blocksPerGrid = getBlocksPerGrid(attribute_size);
	processConceptKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_attr,
			d_attributes, attribute_size);
	hipMemcpy(concept_latice[concept_count].attributes, d_attributes,
			sizeof(int) * attribute_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_attr);
	hipFree(d_attributes);
	// --------------------------------------------------------------------------------------------------------------

	concept_count++;
}

// check attribute contains on attribute list or not
bool checkAttribute(int j, int *attr) {
	bool status = true;
	if (attr[j] == 0) {
		status = false;
	}
	return status;
}

// make extent
void makeExtent(int *extent, int *obj, int attr_index) {
	// >>> Run CUDA Kernel <<<
	int *d_obj;
	int *d_extent;
	hipMalloc((void**) &d_obj, sizeof(int) * data_size);
	hipMalloc((void**) &d_extent, sizeof(int) * data_size);
	hipMemcpy(d_obj, obj, sizeof(int) * data_size, hipMemcpyHostToDevice);
	int blocksPerGrid = getBlocksPerGrid(data_size);
	makeExtentKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_extent, d_obj,
			d_cross_table, attr_index, data_size, attribute_size);
	hipMemcpy(extent, d_extent, sizeof(int) * data_size,
			hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_obj);
	hipFree(d_extent);
}

// make intent
void makeIntent(int *intent, int *extent, int attr_index) {
	int empty_count = 0;

	// check extent is empty set
	// --------------------------------------------------------------------------------------------------------------
	// >>> Run CUDA Kernel <<<
	int *d_extent;
	int *d_empty_count;
	hipMalloc((void**) &d_extent, sizeof(int) * data_size);
	hipMalloc((void**) &d_empty_count, sizeof(int));
	hipMemcpy(d_extent, extent, sizeof(int) * data_size,
			hipMemcpyHostToDevice);
	hipMemcpy(d_empty_count, &empty_count, sizeof(int),
			hipMemcpyHostToDevice);
	int blocksPerGrid = getBlocksPerGrid(data_size);
	makeIntentKernelOnExtentEmpty<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
			d_extent, d_empty_count, data_size);
	hipMemcpy(&empty_count, d_empty_count, sizeof(int),
			hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_empty_count);
	// --------------------------------------------------------------------------------------------------------------

	// --------------------------------------------------------------------------------------------------------------
	// >>> Run CUDA Kernel <<<
	int *d_intent;
	hipMalloc((void**) &d_intent, sizeof(int) * attribute_size);
	blocksPerGrid = getBlocksPerGrid(attribute_size);
	makeIntentKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_intent, d_extent,
			d_cross_table, empty_count, data_size, attribute_size);
	hipMemcpy(intent, d_intent, sizeof(int) * attribute_size,
			hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_intent);
	// --------------------------------------------------------------------------------------------------------------
}

// perform canonicity test
bool canonicity(int *attr, int *intent, int attr_index) {
	bool status = true;
	int set_1[attr_index];
	int set_2[attr_index];
	int set_1_c = 0; // holds set 1 found count
	int set_2_c = 0; // holds set 2 found count

	// 1. check on attribute list
	// --------------------------------------------------------------------------------------------------------------
	// >>> Run CUDA Kernel <<<
	int *d_set_1;
	int *d_set_2;
	int *d_set_1_c;
	int *d_set_2_c;
	int *d_attr;
	int *d_intent;
	hipMalloc((void**) &d_set_1, sizeof(int) * attr_index);
	hipMalloc((void**) &d_set_2, sizeof(int) * attr_index);
	hipMalloc((void**) &d_set_1_c, sizeof(int));
	hipMalloc((void**) &d_set_2_c, sizeof(int));
	hipMalloc((void**) &d_attr, sizeof(int) * attribute_size);
	hipMalloc((void**) &d_intent, sizeof(int) * attribute_size);
	hipMemcpy(d_set_1_c, &set_1_c, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_set_2_c, &set_2_c, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_attr, attr, sizeof(int) * attribute_size,
			hipMemcpyHostToDevice);
	hipMemcpy(d_intent, intent, sizeof(int) * attribute_size,
			hipMemcpyHostToDevice);
	int blocksPerGrid = getBlocksPerGrid(attr_index);
	canonicityKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_set_1, d_set_1_c,
			d_set_2, d_set_2_c, d_attr, d_intent, attr_index);
	hipMemcpy(&set_1_c, d_set_1_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&set_2_c, d_set_2_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(set_1, d_set_1, sizeof(int) * attr_index,
			hipMemcpyDeviceToHost);
	hipMemcpy(set_2, d_set_2, sizeof(int) * attr_index,
			hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // force host to pause until the copying is done
	hipFree(d_set_1_c);
	hipFree(d_set_2_c);
	hipFree(d_attr);
	hipFree(d_intent);
	// --------------------------------------------------------------------------------------------------------------

	if (set_1_c == 0 && set_2_c == 0) {
		// both are empty set
		status = true;
	} else if ((set_1_c != 0 && set_2_c == 0)
			|| (set_1_c == 0 && set_2_c != 0)) {
		// found some element(s) on either of set
		status = false;
	} else if (set_1_c == set_2_c) {
		// found element(s) on both sets
		// ----------------------------------------------------------------------------------------------------------
		// >>> Run CUDA Kernel <<<
		bool *d_status;
		hipMalloc((void**) &d_status, sizeof(bool));
		hipMemcpy(d_status, &status, sizeof(bool), hipMemcpyHostToDevice);
		blocksPerGrid = getBlocksPerGrid(attr_index);
		canonicityKernelSetCheck<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_set_1,
				d_set_2, d_status, attr_index);
		hipMemcpy(&status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
		hipDeviceSynchronize(); // force host to pause until the copying is done
		hipFree(d_status);
		// ----------------------------------------------------------------------------------------------------------
	} else {
		status = false;
	}

	// --------------------------------------------------------------------------------------------------------------
	// clear up device set_1 and set_2
	hipFree(d_set_1);
	hipFree(d_set_2);
	// --------------------------------------------------------------------------------------------------------------

	return status;
}

// Generate Blocks Per Grid for given size
int getBlocksPerGrid(int size) {
	int blocksPerGrid = size / THREADS_PER_BLOCK;
	if ((size % THREADS_PER_BLOCK) != 0) {
		blocksPerGrid++;
	}
	return blocksPerGrid;
}

// ******************************************************************************************************************
// CUDA Kernels
// ******************************************************************************************************************

// CUDA Kernel to process - 1 Loop
__global__ void processConceptKernel(int *set_1, int *set_2, int size) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < size)
		set_2[i] = set_1[i];
}

// CUDA Kernel to make Extent
__global__ void makeExtentKernel(int *extent, int *obj, int *cross_objects,
		int attr_index, int rows, int cols) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < rows) {
		extent[i] = -1;
		if (cross_objects[i * cols + attr_index] == 1 && obj[i] != -1) {
			extent[i] = i;
		}
	}
}

// CUDA Kernel to make Extent Empty count on Intent making
__global__ void makeIntentKernelOnExtentEmpty(int *extent, int *empty_count,
		int size) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < size) {
		if (extent[i] == -1) {
			empty_count++;
		}
	}
}

// CUDA Kernel to make Intent
__global__ void makeIntentKernel(int *intent, int *extent, int *cross_objects,
		int empty_count, int rows, int cols) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < cols) {
		if (empty_count != rows) {
			bool status = true;
			int i;
			for (i = 0; i < rows; i++) {
				// check extent available
				if (extent[i] != -1) {
					// check related cross table index of current exten attribute availability
					if (cross_objects[i * cols + index] != 1) {
						status = false;
						break;
					}
				}
			}
			if (status) {
				intent[index] = 1;
			} else {
				intent[index] = 0;
			}
		} else {
			intent[index] = 1;
		}
	}
}

// CUDA Kernel to proccess canonicity test
__global__ void canonicityKernel(int *set_1, int *set_1_c, int *set_2,
		int *set_2_c, int *attr, int *intent, int cols) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < cols) {
		// check attr set
		if (attr[i] == 1) {
			*set_1_c = *set_1_c + 1;
			set_1[i] = 1;
		} else {
			set_1[i] = 0;
		}

		// check intent set
		if (intent[i] == 1) {
			*set_2_c = *set_2_c + 1;
			set_2[i] = 1;
		} else {
			set_2[i] = 0;
		}
	}
}

// CUDA Kernel to check canonicity test set_1 and set_2 sets data
__global__ void canonicityKernelSetCheck(int *set_1, int *set_2, bool *status,
		int cols) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < cols && *status) {
		if (set_1[i] != set_2[i]) {
			*status = false;
		}
	}
}

// ******************************************************************************************************************
